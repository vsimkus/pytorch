#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/native/cuda/zmath.cuh>

namespace at { namespace native {

void bitwise_not_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a) {
      return !a;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ~a;
      });
    });
  }
}

void expm1_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "expm1_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::expm1(a);
    });
  });
}

// We manually overload rsqrt because std::rsqrt does not work with thrust::complex types.
template<typename scalar_t>
__host__ __device__ static inline scalar_t rsqrt_wrapper(scalar_t v) {
  return ::rsqrt(v);
}

template<typename T>
__host__ __device__ static inline thrust::complex<T> rsqrt_wrapper(thrust::complex<T> v) {
  const thrust::complex<T> one = thrust::complex<T>(1.0, 0);
  return one/thrust::sqrt(v);
}

void rsqrt_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "rsqrt_cuda", [&]() {
    using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
    gpu_kernel(iter, []GPU_LAMBDA(thrust_t a) -> thrust_t {
      // In CUDA, ::rsqrt is overloaded for float and at::Half here is implicitly cast to float.
      return rsqrt_wrapper(a);
    });
  });
}

// We manually overload sqrt because std::sqrt does not work with thrust::complex types.
template<typename scalar_t>
__host__ __device__ static inline scalar_t sqrt_wrapper(scalar_t v) {
  return ::sqrt(v);
}

template<typename T>
__host__ __device__ static inline thrust::complex<T> sqrt_wrapper(thrust::complex<T> v) {
  return thrust::sqrt(v);
}

void sqrt_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "sqrt_cuda", [&]() {
    using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
    gpu_kernel(iter, []GPU_LAMBDA(thrust_t a) -> thrust_t {
      return sqrt_wrapper(a);
    });
  });
}

void sigmoid_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "sigmoid_cuda", [&]() {
    AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "sigmoid_cuda", [&] {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        scalar_t one = scalar_t(1);
        return  one / (one + std::exp(- a));
      });
    });
  });
}

void erfinv_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "erfinv_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erfinv(a);
    });
  });
}

template<typename scalar_t>
__host__ __device__ static inline scalar_t zabs_cuda_wrapper(scalar_t v) {
  return v;
}

template<typename T>
__host__ __device__ static inline T zabs_cuda_wrapper(thrust::complex<T> v) {
  return thrust::abs(v);
}

void clamp_kernel_cuda(TensorIterator& iter, Scalar min_scalar, Scalar max_scalar) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND(ScalarType::Half, iter.dtype(), "clamp_cuda", [&]() {
    using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
    auto min = thrust_t(min_scalar.to<scalar_t>());
    auto max = thrust_t(max_scalar.to<scalar_t>());
    gpu_kernel(iter,
      [=]GPU_LAMBDA(thrust_t a) -> thrust_t {
        return ((zabs_cuda_wrapper(a) < zabs_cuda_wrapper(min)) ?
                min : ((zabs_cuda_wrapper(a) > zabs_cuda_wrapper(max)) ? max : a));
    });
  });
}

void clamp_max_kernel_cuda(TensorIterator& iter, Scalar max_scalar) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND(ScalarType::Half, iter.dtype(), "clamp_max_cuda", [&]() {
    using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
    auto max = thrust_t(max_scalar.to<scalar_t>());
    gpu_kernel(iter,
      [=]GPU_LAMBDA(thrust_t a) -> thrust_t {
       return (zabs_cuda_wrapper(a) > zabs_cuda_wrapper(max)) ? max : a;
    });
  });
}

void clamp_min_kernel_cuda(TensorIterator& iter, Scalar min_scalar) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND(ScalarType::Half, iter.dtype(), "clamp_min_cuda", [&]() {
    using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
    auto min = thrust_t(min_scalar.to<scalar_t>());
    gpu_kernel(iter,
      [=]GPU_LAMBDA(thrust_t a) -> thrust_t {
       return (zabs_cuda_wrapper(a) < zabs_cuda_wrapper(min)) ? min : a;
    });
  });
}

void clamp_with_tensors_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND(ScalarType::Half, iter.dtype(), "clamp_cuda", [&]() {
    using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
    gpu_kernel(iter,
      [=]GPU_LAMBDA(thrust_t a, thrust_t min, thrust_t max) -> thrust_t {
       return ((zabs_cuda_wrapper(a) < zabs_cuda_wrapper(min)) ?
                min : ((zabs_cuda_wrapper(a) > zabs_cuda_wrapper(max)) ? max : a));
    });
  });
}

void clamp_max_with_tensor_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND(ScalarType::Half, iter.dtype(), "clamp_max_cuda", [&]() {
    using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
    gpu_kernel(iter,
      [=]GPU_LAMBDA(thrust_t a, thrust_t max) -> thrust_t {
       return (zabs_cuda_wrapper(a) > zabs_cuda_wrapper(max)) ? max : a;
    });
  });
}

void clamp_min_with_tensor_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND(ScalarType::Half, iter.dtype(), "clamp_min_cuda", [&]() {
    using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
    gpu_kernel(iter,
      [=]GPU_LAMBDA(thrust_t a, thrust_t min) -> thrust_t {
       return (zabs_cuda_wrapper(a) < zabs_cuda_wrapper(min)) ? min : a;
    });
  });
}

REGISTER_DISPATCH(bitwise_not_stub, &bitwise_not_kernel_cuda);
REGISTER_DISPATCH(expm1_stub, &expm1_kernel_cuda);
REGISTER_DISPATCH(rsqrt_stub, &rsqrt_kernel_cuda);
REGISTER_DISPATCH(sqrt_stub, &sqrt_kernel_cuda);
REGISTER_DISPATCH(sigmoid_stub, &sigmoid_kernel_cuda);
REGISTER_DISPATCH(erfinv_stub, &erfinv_kernel_cuda);
REGISTER_DISPATCH(clamp_stub, &clamp_kernel_cuda);
REGISTER_DISPATCH(clamp_max_stub, &clamp_max_kernel_cuda);
REGISTER_DISPATCH(clamp_min_stub, &clamp_min_kernel_cuda);
REGISTER_DISPATCH(clamp_with_tensors_stub, &clamp_with_tensors_kernel_cuda);
REGISTER_DISPATCH(clamp_max_with_tensor_stub, &clamp_max_with_tensor_kernel_cuda);
REGISTER_DISPATCH(clamp_min_with_tensor_stub, &clamp_min_with_tensor_kernel_cuda);
}}
